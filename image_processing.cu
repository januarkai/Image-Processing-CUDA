#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "image_processing.hpp"

// Indexing function for 2D image + RGB Channels
__device__ int index2D(int Channels) {
	int blockId = blockIdx.x + blockIdx.y * gridDim.x;
	int idx = (blockId * (blockDim.x * blockDim.y)+ (threadIdx.y * blockDim.x)+ threadIdx.x) * Channels;
	return idx;
}


// Kernel function to run image grayscle computations
__global__ void grayScale_kernel(unsigned char* image, int channels, unsigned char* output){

	int idx = index2D(channels);
	int r, g, b, temp;

	b = image[idx];
	g = image[idx + 1];
	r = image[idx + 2];
	
	temp = (b+g+r)/3;

	output[idx/ channels] = temp;

}


// CPU function to initialize CUDA kernel and memory copy 
void grayScaleCUDA(unsigned char* input_image, int height, int width, int channels, unsigned char* output_image){
    
    unsigned char* dev_input_image = NULL;
	unsigned char* dev_output_image = NULL;
	int size = sizeof(int);

	hipMalloc((void**)&dev_input_image, height * width * channels);
	hipMalloc((void**)&dev_output_image, height * width * channels);

	hipMemcpy(dev_input_image, input_image, height * width * channels, hipMemcpyHostToDevice);
	hipMemcpy(dev_output_image, output_image, height * width, hipMemcpyHostToDevice);


	dim3 Thread_Index(32, 32);
	dim3 Grid_Image(width/Thread_Index.x+1, height/Thread_Index.y+1);
	grayScale_kernel<<<Grid_Image, Thread_Index>>>(dev_input_image, channels, dev_output_image);

	hipMemcpy(output_image, dev_output_image, height * width, hipMemcpyDeviceToHost);

	hipDeviceSynchronize();

	hipFree(dev_input_image);
	hipFree(dev_output_image);
}

unsigned char* createImageBuffer(unsigned int bytes) {

    unsigned char *ptr = NULL;
    hipSetDeviceFlags(hipDeviceMapHost);
    hipHostAlloc(&ptr, bytes, hipHostMallocMapped);
    return ptr;
}

void destroyImageBuffer(unsigned char* bytes) {
    hipHostFree(bytes);
}